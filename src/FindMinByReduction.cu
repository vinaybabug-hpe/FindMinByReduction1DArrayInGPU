/*
 ============================================================================
 Name        : FindMinByReduction.cu
 Author      : Vinay B Gavirangaswamy
 Version     :
 Copyright   : Put copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>

#if __DEVICE_EMULATION__
#define DEBUG_SYNC __syncthreads();
#else
#define DEBUG_SYNC
#endif

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

#ifndef MIN_IDX
#define MIN_IDX(x,y, idx_x, idx_y) ((x < y) ? idx_x : idx_y)
#endif

#if (__CUDA_ARCH__ < 200)
#define int_mult(x,y)	__mul24(x,y)
#else
#define int_mult(x,y)	x*y
#endif

#define inf 0x7f800000

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory {
	__device__ inline operator T *() {
		extern __shared__ int __smem[];
		return (T *) __smem;
	}

	__device__ inline operator const T *() const {
		extern __shared__ int __smem[];
		return (T *) __smem;
	}
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double> {
	__device__ inline operator double *() {
		extern __shared__ double __smem_d[];
		return (double *) __smem_d;
	}

	__device__ inline operator const double *() const {
		extern __shared__ double __smem_d[];
		return (double *) __smem_d;
	}
};



/*
 This version finds minimum and index at which it was found in multiple elements per thread sequentially.  This reduces the overall
 cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
 (Brent's Theorem optimization)

 Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
 In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
 If blockSize > 32, allocate blockSize*sizeof(T) bytes.
 */
template<class T, unsigned int blockSize, bool nIsPow2>
__global__ void reduceMin6(T *g_idata, int *g_idxs, T *g_odata, int *g_oIdxs, unsigned int n) {

	T *sdata = SharedMemory<T>();
	int *sdataIdx = ((int *)sdata) + blockSize;



	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;


	T myMin = 99999;
	int myMinIdx = -1;
	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n) {
		myMinIdx  = MIN_IDX(g_idata[i], myMin, g_idxs[i], myMinIdx);
		myMin = MIN(g_idata[i], myMin);



		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n){
			//myMin += g_idata[i + blockSize];
			myMinIdx  = MIN_IDX(g_idata[i + blockSize], myMin, g_idxs[i + blockSize], myMinIdx);
			myMin = MIN(g_idata[i + blockSize], myMin);
		}

		i += gridSize;
	}


	// each thread puts its local sum into shared memory
	sdata[tid] = myMin;
	sdataIdx[tid] = myMinIdx;

	__syncthreads();

	// do reduction in shared mem
	if ((blockSize >= 512) && (tid < 256)) {

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 256], myMin, sdataIdx[tid + 256], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 256], myMin);

	}

	__syncthreads();

	if ((blockSize >= 256) && (tid < 128)) {

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 128], myMin, sdataIdx[tid + 128], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 128], myMin);


	}

	__syncthreads();

	if ((blockSize >= 128) && (tid < 64)) {

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 64], myMin, sdataIdx[tid + 64], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 64], myMin);
	}

	__syncthreads();

#if (__CUDA_ARCH__ >= 300 )
	if (tid < 32) {
		// Fetch final intermediate sum from 2nd warp
		if (blockSize >= 64){

			myMinIdx = MIN_IDX(sdata[tid + 32], myMin, sdataIdx[tid + 32], myMinIdx);
			myMin = MIN(sdata[tid + 32], myMin);
		}
		// Reduce final warp using shuffle
		for (int offset = warpSize / 2; offset > 0; offset /= 2) {

			int tempMyMinIdx = __shfl_down(myMinIdx, offset);
			float tempMyMin = __shfl_down(myMin, offset);

			myMinIdx = MIN_IDX(tempMyMin, myMin, tempMyMinIdx , myMinIdx);
			myMin = MIN(tempMyMin, myMin);

		}

	}
#else
	// fully unroll reduction within a single warp
	if ((blockSize >= 64) && (tid < 32))
	{
		//sdata[tid] = myMin = myMin + sdata[tid + 32];
		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 32], myMin, sdataIdx[tid + 32], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 32], myMin);
	}

	__syncthreads();

	if ((blockSize >= 32) && (tid < 16))
	{


		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 16], myMin, sdataIdx[tid + 16], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 16], myMin);
	}

	__syncthreads();

	if ((blockSize >= 16) && (tid < 8))
	{

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 8], myMin, sdataIdx[tid + 8], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 8], myMin);
	}

	__syncthreads();

	if ((blockSize >= 8) && (tid < 4))
	{

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 4], myMin, sdataIdx[tid + 4], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 4], myMin);
	}

	__syncthreads();

	if ((blockSize >= 4) && (tid < 2))
	{

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 2], myMin, sdataIdx[tid + 2], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 2], myMin);
	}

	__syncthreads();

	if ((blockSize >= 2) && ( tid < 1))
	{

		sdataIdx[tid] = myMinIdx = MIN_IDX(sdata[tid + 1], myMin, sdataIdx[tid + 1], myMinIdx);
		sdata[tid] = myMin = MIN(sdata[tid + 1], myMin);
	}

	__syncthreads();
#endif

	__syncthreads();
	// write result for this block to global mem
	if (tid == 0){
		g_odata[blockIdx.x] = myMin;
		g_oIdxs[blockIdx.x] = myMinIdx;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel
// 6, we observe the maximum specified number of blocks, because each thread in
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks,
		int maxThreads, int &blocks, int &threads) {

	//get device capability, to avoid block/grid size exceed the upper bound
	hipDeviceProp_t prop;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	if (whichKernel < 3) {
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = (n + threads - 1) / threads;
	} else {
		threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
		blocks = (n + (threads * 2 - 1)) / (threads * 2);
	}

	if ((float) threads * blocks
			> (float) prop.maxGridSize[0] * prop.maxThreadsPerBlock) {
		printf("n is too large, please choose a smaller number!\n");
	}

	if (blocks > prop.maxGridSize[0]) {
		printf(
				"Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
				blocks, prop.maxGridSize[0], threads * 2, threads);

		blocks /= 2;
		threads *= 2;
	}

	if (whichKernel == 6) {
		blocks = MIN(maxBlocks, blocks);
	}
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template<class T>
void reduceMin(int size, int threads, int blocks, int whichKernel, T *d_idata,
		T *d_odata, int *idxs, int *oIdxs) {
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize =
	        (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
	smemSize += threads*sizeof(int);

	if (isPow2(size)) {
		switch (threads) {
		case 512:
			reduceMin6<T, 512, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 256:
			reduceMin6<T, 256, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 128:
			reduceMin6<T, 128, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 64:
			reduceMin6<T, 64, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 32:
			reduceMin6<T, 32, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 16:
			reduceMin6<T, 16, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 8:
			reduceMin6<T, 8, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 4:
			reduceMin6<T, 4, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 2:
			reduceMin6<T, 2, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 1:
			reduceMin6<T, 1, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;
		}
	} else {
		switch (threads) {
		case 512:
			reduceMin6<T, 512, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 256:
			reduceMin6<T, 256, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 128:
			reduceMin6<T, 128, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 64:
			reduceMin6<T, 64, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 32:
			reduceMin6<T, 32, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 16:
			reduceMin6<T, 16, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 8:
			reduceMin6<T, 8, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 4:
			reduceMin6<T, 4, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 2:
			reduceMin6<T, 2, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;

		case 1:
			reduceMin6<T, 1, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, idxs,
					d_odata, oIdxs, size);
			break;
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
//! Compute minimum reduction on CPU
//! We use Kahan summation for an accurate sum of large arrays.
//! http://en.wikipedia.org/wiki/Kahan_summation_algorithm
//!
//! @param data       pointer to input data
//! @param size       number of input data elements
//! @param min        minimum value (out)
//! @param min        minimum value index (out)
////////////////////////////////////////////////////////////////////////////////
template<class T>
void reduceMINCPU(T *data, int size, T *min, int *idx)
{
    *min = data[0];
    int min_idx = 0;
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i];
        T t = MIN(*min, y);
        min_idx = MIN_IDX(*min, y, min_idx, i);
        (*min) = t;
    }

    *idx = min_idx;

    return;
}


// Instantiate the reduction function for 3 types
template void
reduceMin<int>(int size, int threads, int blocks, int whichKernel, int *d_idata,
		int *d_odata, int *idxs, int *oIdxs);

template void
reduceMin<float>(int size, int threads, int blocks, int whichKernel, float *d_idata,
		float *d_odata, int *idxs, int *oIdxs);

template void
reduceMin<double>(int size, int threads, int blocks, int whichKernel, double *d_idata,
		double *d_odata, int *idxs, int *oIdxs);

unsigned long long int minimizationViaReduction(int num_els) {


	unsigned long long int delta;

	int maxThreads = 256;  // number of threads per block
	int whichKernel = 6;
	int maxBlocks = 64;

	float* d_in = NULL;
	float* d_out = NULL;
	int *d_idxs = NULL;
	int *d_oIdxs = NULL;

	printf("%d elements\n", num_els);
	printf("%d threads (max)\n", maxThreads);

	int numBlocks = 0;
	int numThreads = 0;
	getNumBlocksAndThreads(whichKernel, num_els, maxBlocks, maxThreads, numBlocks,
			numThreads);


	hipMalloc((void **) &d_in, num_els * sizeof(float));
	hipMalloc((void **) &d_idxs, num_els * sizeof(int));
	hipMalloc((void **) &d_out, numBlocks * sizeof(float));
	hipMalloc((void **) &d_oIdxs, numBlocks * sizeof(int));

	float* in = (float*) malloc(num_els * sizeof(float));
	int *idxs = (int*) malloc(num_els * sizeof(int));
	float* out = (float*) malloc(numBlocks * sizeof(float));
	int* oIdxs = (int*) malloc(numBlocks * sizeof(int));

	for (int i = 0; i < num_els; i++) {
		in[i] = (double) rand() / (double) RAND_MAX;
		idxs[i] = i;
	}


	// copy data directly to device memory
	hipMemcpy(d_in, in, num_els * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_idxs, idxs, num_els * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, numBlocks * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_oIdxs, oIdxs, numBlocks * sizeof(int),hipMemcpyHostToDevice);

	reduceMin<float>(num_els, numThreads, numBlocks, whichKernel, d_in, d_out, d_idxs, d_oIdxs);

	hipMemcpy(out, d_out, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(oIdxs, d_oIdxs, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

	int min_idx = -1;
	float min = 999999;

	for(int i=0; i< numBlocks; i++){

		printf("\n Reduce MIN \ BLOCK GPU idx: %d  value: %f", oIdxs[i], out[i]);
		min_idx = MIN_IDX(out[i], min, oIdxs[i], min_idx);
		min = MIN(out[i], min);

	}


	printf("\n\n Reduce MIN GPU idx: %d  value: %f\n", min_idx, min);


	reduceMINCPU<float>(in, num_els, &min, &min_idx);


	printf("\n\n Reduce MIN CPU idx: %d  value: %f", min_idx, min);

	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_idxs);

	free(in);
	free(out);

	//system("pause");

	return delta;

}

int main(int argc, char* argv[]) {

	minimizationViaReduction(1024);

	return 0;
}
